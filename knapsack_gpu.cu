
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

struct TItem {
    int price;
    int weight;
    bool operator<(const TItem& other) const {
        return (double)price / weight > (double)other.price / other.weight;
    }
};

const int THREADS_PER_BLOCK = 192;

void BranchCPU(ssize_t e, int* w, int* p, int* s, int* U_old, int k, int* weight, int* price) {
    int s_e = s[e];
    if (k < s_e) {
        w[e] -= weight[k];
        p[e] -= price[k];
    } else {
        ++s[e];
        U_old[e] = 0;
    }
}

__global__ void BranchGPU(int* w, int* p, int* s, int* U_old, int k, int* weight, int* price, ssize_t q) {
    ssize_t e = blockIdx.x * blockDim.x + threadIdx.x;
    if (e >= q) {
    	return;
    }
    int s_e = s[e];
    if (k < s_e) {
        w[e] -= weight[k];
        p[e] -= price[k];
    } else {
        ++s[e];
        U_old[e] = 0;
    }
}

void BoundCPU(ssize_t e, int* w, int* p, int* s, int* L, int* U, int k, int n, int W, int* weight, int* price) {
    int i = s[e], w_e = w[e], p_e = p[e], weight_i = 0, price_i = 0;
    for (; i <= n; ++i) {
        weight_i = weight[i];
        price_i = price[i];
        if (w_e + weight_i <= W) {
            w_e += weight_i;
            p_e += price_i;
        } else {
            break;
        }
    }
    U[e] = p_e + (weight_i ? (W - w_e) * price_i / weight_i : 0);
    w[e] = w_e;
    p[e] = p_e;
    s[e] = i;

    for (; i < n; ++i) {
        weight_i = weight[i];
        price_i = price[i];
        if (w_e + weight_i <= W) {
            w_e += weight_i;
            p_e += price_i;
        }
    }
    L[e] = p_e;
}

__global__ void BoundGPU(int* w, int* p, int* s, int* L, int* U, int k, int n, int W, int* weight, int* price, ssize_t q) {
    ssize_t e = blockIdx.x * blockDim.x + threadIdx.x;
    if (e >= q) {
    	return;
    }
    int i = s[e], w_e = w[e], p_e = p[e], weight_i = 0, price_i = 0;
    for (; i <= n; ++i) {
        weight_i = weight[i];
        price_i = price[i];
        if (w_e + weight_i <= W) {
            w_e += weight_i;
            p_e += price_i;
        } else {
            break;
        }
    }
    U[e] = p_e + (weight_i ? (W - w_e) * price_i / weight_i : 0);
    w[e] = w_e;
    p[e] = p_e;
    s[e] = i;

    for (; i < n; ++i) {
        weight_i = weight[i];
        price_i = price[i];
        if (w_e + weight_i <= W) {
            w_e += weight_i;
            p_e += price_i;
        }
    }
    L[e] = p_e;
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " input_file output_file" << std::endl;
        return 0;
    }
    std::ifstream fin(argv[1]);
    std::ofstream fout(argv[2]);
    int n, W;
    fin >> n >> W;
    std::vector<TItem> items(n);
    for (int i = 0; i < n; ++i) {
        fin >> items[i].price >> items[i].weight;
    }
    std::sort(items.begin(), items.end());

    int* weight = (int*)malloc((n + 1) * sizeof(*weight));
    int* price = (int*)malloc((n + 1) * sizeof(*price));
    for (int i = 0; i < n; ++i) {
        weight[i] = items[i].weight;
        price[i] = items[i].price;
    }
    weight[n] = price[n] = 0;

    std::chrono::high_resolution_clock::time_point total_start = std::chrono::high_resolution_clock::now();

    int *cuda_weight = nullptr, *cuda_price = nullptr;
    ssize_t q = 1;
    int* w = (int*)malloc(q * sizeof(*w));
    int* p = (int*)malloc(q * sizeof(*p));
    int* s = (int*)malloc(q * sizeof(*s));
    int* L = (int*)malloc(q * sizeof(*L));
    int* U = (int*)malloc(q * sizeof(*U));
    w[0] = p[0] = s[0]= 0;

    BoundCPU(0, w, p, s, L, U, 0, n, W, weight, price);
    int record = L[0];
    free(L);

    for (int k = 0; k < n; ++k) {
        std::cout << "Step " << k + 1 << ", q = " << q << std::endl;
        if (q > 5000000) {
            if (cuda_weight == nullptr) {
                hipMalloc(&cuda_weight, (n + 1) * sizeof(*cuda_weight));
                hipMalloc(&cuda_price, (n + 1) * sizeof(*cuda_price));
                hipMemcpy(cuda_weight, weight, (n + 1) * sizeof(*cuda_weight), hipMemcpyHostToDevice);
                hipMemcpy(cuda_price, price, (n + 1) * sizeof(*cuda_price), hipMemcpyHostToDevice);
            }
            int *w_new, *p_new, *s_new, *L_new, *U_new, *U_old;
            hipMalloc(&w_new, q * sizeof(*w_new));
            hipMalloc(&p_new, q * sizeof(*p_new));
            hipMalloc(&s_new, q * sizeof(*s_new));
            hipMalloc(&U_old, q * sizeof(*U_old));
            hipMemcpy(w_new, w, q * sizeof(*w), hipMemcpyHostToDevice);
            hipMemcpy(p_new, p, q * sizeof(*p), hipMemcpyHostToDevice);
            hipMemcpy(s_new, s, q * sizeof(*s), hipMemcpyHostToDevice);
            hipMemcpy(U_old, U, q * sizeof(*U), hipMemcpyHostToDevice);

            const ssize_t q_block = (q + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            BranchGPU<<<q_block, THREADS_PER_BLOCK>>>(w_new, p_new, s_new, U_old, k, cuda_weight, cuda_price, q);
            hipDeviceSynchronize();
            
            hipMemcpy(U, U_old, q * sizeof(*U), hipMemcpyDeviceToHost);
            hipFree(U_old);
            hipMalloc(&L_new, q * sizeof(*L_new));
            hipMalloc(&U_new, q * sizeof(*U_new));

            BoundGPU<<<q_block, THREADS_PER_BLOCK>>>(w_new, p_new, s_new, L_new, U_new, k, n, W, cuda_weight, cuda_price, q);
            hipDeviceSynchronize();

            int *L_new_CPU = (int*)malloc(q * sizeof(*L_new_CPU));
            hipMemcpy(L_new_CPU, L_new, q * sizeof(*L_new), hipMemcpyDeviceToHost);
            hipFree(L_new);
            for (ssize_t e = 0; e < q; ++e) {
                record = std::max(record, L_new_CPU[e]);
            }
            free(L_new_CPU);

            w = (int*)realloc(w, 2 * q * sizeof(*w));
            p = (int*)realloc(p, 2 * q * sizeof(*p));
            s = (int*)realloc(s, 2 * q * sizeof(*s));
            U = (int*)realloc(U, 2 * q * sizeof(*U));
            hipMemcpy(w + q, w_new, q * sizeof(*w), hipMemcpyDeviceToHost);
            hipMemcpy(p + q, p_new, q * sizeof(*p), hipMemcpyDeviceToHost);
            hipMemcpy(s + q, s_new, q * sizeof(*s), hipMemcpyDeviceToHost);
            hipMemcpy(U + q, U_new, q * sizeof(*U), hipMemcpyDeviceToHost);
            hipFree(w_new);
            hipFree(p_new);
            hipFree(s_new);
            hipFree(U_new);

        } else {

            w = (int*)realloc(w, 2 * q * sizeof(*w));
            p = (int*)realloc(p, 2 * q * sizeof(*p));
            s = (int*)realloc(s, 2 * q * sizeof(*s));
            memcpy(w + q, w, q * sizeof(*w));
            memcpy(p + q, p, q * sizeof(*p));
            memcpy(s + q, s, q * sizeof(*s));
            for (ssize_t e = 0; e < q; ++e) {
                BranchCPU(e, w + q, p + q, s + q, U, k, weight, price);
            }

            U = (int*)realloc(U, 2 * q * sizeof(*U));
            int* L_new = (int*)malloc(q * sizeof(*L_new));
            for (ssize_t e = 0; e < q; ++e) {
                BoundCPU(e, w + q, p + q, s + q, L_new, U + q, k, n, W, weight, price);
                record = std::max(record, L_new[e]);
            }
            free(L_new);
        }

        for (ssize_t i = 0, j = 2 * q - 1; ;) {
            while (i < 2 * q && U[i] >= record) {
                ++i;
            }
            while (j >= 0 && U[j] < record) {
                --j;
            }
            if (i >= j) {
                q = i;
                break;
            }
            w[i] = w[j];
            p[i] = p[j];
            s[i] = s[j];
            std::swap(U[i], U[j]);
        }
        if (q == 0) {
            break;
        }
    }

    free(w);
    free(p);
    free(s);
    free(U);
    free(weight);
    free(price);
    if (cuda_weight != nullptr) {
        hipFree(cuda_weight);
        hipFree(cuda_price);
    }

    std::chrono::high_resolution_clock::time_point total_end = std::chrono::high_resolution_clock::now();
    double total_time = std::chrono::duration_cast<std::chrono::duration<double>>(total_end - total_start).count();
    std::cout << "Total time: " << total_time << std::endl;
    fout << record << std::endl;

    return 0;
}
